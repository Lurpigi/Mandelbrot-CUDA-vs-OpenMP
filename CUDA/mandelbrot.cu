#include <iostream>
#include <fstream>
#include <complex>
#include <chrono>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include <hip/hip_complex.h>

// Ranges of the set
#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1

// Image ratio
#define RATIO_X (MAX_X - MIN_X)
#define RATIO_Y (MAX_Y - MIN_Y)

// Image size
#define RESOLUTION 3000 //1000
#define WIDTH (RATIO_X * RESOLUTION)
#define HEIGHT (RATIO_Y * RESOLUTION)

#define STEP ((double)RATIO_X / WIDTH)

#define DEGREE 2        // Degree of the polynomial
#define ITERATIONS 1000 // Maximum number of iterations

using namespace std;


__global__ void mandelbrot(int *image)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (j > 0 && j <=HEIGHT - 1 && i > 0 && i <= WIDTH - 1){
        int pos = j * WIDTH + i;

        image[pos] = 0;
        hipDoubleComplex c = make_hipDoubleComplex(i * STEP + MIN_X, j * STEP + MIN_Y);

        hipDoubleComplex z = make_hipDoubleComplex(0.0, 0.0);
        for (int k = 1; k <= ITERATIONS; k++)
        {
            z = hipCadd(hipCmul(z, z), c);
            if (hipCabs(z) >= 2) {
                image[pos] = k;
                break;
            }
        }

    }

        
    
}

void cuda_check(hipError_t err)
{
  if (err != hipSuccess){
    fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    exit(err);
  }
}

int main(int argc, char **argv)
{

    if(argc < 2)
    {
        cout << "Usage: "<<argv[0] <<" THREAD_SIDE" << endl;
        return -1;
    }

    int THREAD_SIDE = atoi(argv[1]);

    int *const image = new int[HEIGHT * WIDTH];

    int* image_device;
    cuda_check(hipMalloc(&image_device, HEIGHT * WIDTH * sizeof(int)));

    struct timeval t1, t2;

    gettimeofday(&t1, 0);


    dim3 threads(THREAD_SIDE, THREAD_SIDE);
    dim3 blocks((WIDTH + threads.x - 1) / threads.x, (HEIGHT + threads.y - 1) / threads.y);

    // Call the kernel
    mandelbrot<<<blocks,threads>>>(image_device);

    cuda_check(hipMemcpy(image, image_device, HEIGHT * WIDTH * sizeof(int), hipMemcpyDeviceToHost));
    cuda_check(hipFree(image_device));

    gettimeofday(&t2, 0);

    cout << "Time elapsed: " << (t2.tv_sec - t1.tv_sec) + (t2.tv_usec - t1.tv_usec) * 0.000001 << " seconds." << endl;
    cout << "<<" << blocks.x <<" x "<< blocks.y <<","<< threads.x <<" x "<< threads.y << ">>" << endl;

    // Write the result to a file
    ofstream matrix_out;

    if (argc < 3)
    {
        cout << "Please specify the output file as a parameter." << endl;
        return -1;
    }

    matrix_out.open(argv[2], ios::trunc);
    if (!matrix_out.is_open())
    {
        cout << "Unable to open file." << endl;
        return -2;
    }

    for (int row = 0; row < HEIGHT; row++)
    {
        for (int col = 0; col < WIDTH; col++)
        {
            matrix_out << image[row * WIDTH + col];

            if (col < WIDTH - 1)
                matrix_out << ',';
        }
        if (row < HEIGHT - 1)
            matrix_out << endl;
    }
    matrix_out.close();

    delete[] image; // It's here for coding style, but useless
    return 0;
}